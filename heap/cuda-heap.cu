#include "hip/hip_runtime.h"
    #include <cstdio>        // Added for printf() function 
    #include <sys/time.h>    // Added to get time of day
    #include <hip/hip_runtime.h>
    #include <fstream>
    #include <time.h>
    #include <iostream>
    #include <thrust/host_vector.h>
    #include <thrust/device_vector.h>
    #include <thrust/sort.h>
    #include <thrust/iterator/reverse_iterator.h>

    using namespace std;
    //total size of the heap
    #define maxSize 100000000
#define maxSizeDelete 10000000
#define NRUNS 5

    // __global__ void delete_Elem(int *heap,int *d_elements,int *curSize,int *elemSize,int k){
       
    // }

    int getRandom(int lower, int upper)
    {
        int num = (rand() % (upper - lower + 1)) + lower;
        return num;  
    }

    void printArray(thrust::host_vector<int> &arr,int size)
    {
        for(int i = 0;i<size;i++)
            printf("%d, ",arr[i]);
    }

    void FillArray(thrust::host_vector<int> &elements,int size)
    {
        for(int i = 0;i<size;i++)
        {
            elements[i] = getRandom(1,maxSize*10);
        }
    }

    double rtclock(){
        struct timezone Tzp;
        struct timeval Tp;
        int stat;
        stat = gettimeofday(&Tp, &Tzp);
        if (stat != 0) printf("Error return from gettimeofday: %d", stat);
        return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
    }

    void printtime(const char *str, double starttime, double endtime){
        printf("%s%3f seconds\n", str, endtime - starttime);
    }

    //Insert If only key is there
    __global__ void Insert_Elem(int *heap,int *curSize,int *d_elements,int *elemSize)
    {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if(tid < *elemSize)
        {
            heap[tid + *curSize] = d_elements[tid];
        }
    }

    //Insert if both key and values are there
    __global__ void Insert_Elem(int *d_val, int *heap_val,int *heap,int *curSize,int *d_elements,int *elemSize)
    {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if(tid < *elemSize)
        {
            heap[tid + *curSize] = d_elements[tid];
            heap_val[tid + *curSize] = d_val[tid];
        }
    }

    class Heap{
        private:
        int *heap,*heap_val,*curSize;
        bool isSorted;
        int isType;
        public:
        Heap(){
            srand(time(0));
            // cout << "Constructor Called" << endl;
            hipHostAlloc(&curSize, sizeof(int), 0);
            hipMalloc(&heap,maxSize*sizeof(int));
            hipMalloc(&heap_val,maxSize*sizeof(int));
            isSorted = false;
            isType = 0;
            // *curSize = 0;
        }

        int getSize(){
            return *curSize;
        }

        //Insert If only key is there
        void insert(thrust::host_vector<int> &elements,int size1){
            isType = 1;
            isSorted = false;
            thrust::device_vector<int> d_el = elements;
            int *elemSize;
            hipMalloc(&elemSize,sizeof(int));
            hipMemcpy(elemSize,&size1,sizeof(int),hipMemcpyHostToDevice);
            
            int *d_elements = thrust::raw_pointer_cast(d_el.data());

            int block = ceil((float) size1/1024);
            Insert_Elem<<<block,1024>>>(heap,curSize,d_elements,elemSize);
            hipDeviceSynchronize();
            *curSize = *curSize + size1; 
            // printArray(heap,*curSize);
        }

        //Insert if both key and values are there
        void insert(thrust::host_vector<int> &elements,thrust::host_vector<int> &val,int size1){
            isType = 2;
            isSorted = false;
            thrust::device_vector<int> d_el = elements;
            thrust::device_vector<int> d_v = val;
            int *elemSize;
            
            int *d_elements = thrust::raw_pointer_cast(d_el.data());
            int *d_val = thrust::raw_pointer_cast(d_v.data());
            hipMalloc(&elemSize,sizeof(int));
            hipMemcpy(elemSize,&size1,sizeof(int),hipMemcpyHostToDevice);
            
            int block = ceil((float) size1/1024);
            Insert_Elem<<<block,1024>>>(d_val,heap_val,heap,curSize,d_elements,elemSize);
            hipDeviceSynchronize();
            *curSize = *curSize + size1; 
        }


        thrust::host_vector<int>  deleteElem(){
            return deleteElem(1);
        }

        // 1 2 3 4 5 6 7 8 9 10
        thrust::host_vector<int>  deleteElem(int n){
            
            //wrap raw pointer with a device_ptr
            thrust::device_ptr<int> d_vec(heap);
            thrust::device_ptr<int> d_values(heap_val);

            //use device_ptr in thrust algorithms
            if(isSorted == false)
            {
                if(isType == 2)
                    thrust::sort_by_key(d_vec, d_vec+*curSize,d_values,thrust::greater<int>());
                else if(isType == 1)
                    thrust::sort(d_vec, d_vec+*curSize,thrust::greater<int>());

            }
            
            isSorted = true;

            /*cout << endl<<"Array after sorting"<<endl;
            for(int i = 0;i<*curSize;i++){
                cout << d_vec[i] << "->"<< d_values[i] << " ; ";
            }
            cout << endl;
		*/

            thrust::host_vector<int> ret_elements(n);
            if(isType == 2)
                ret_elements.resize(2*n);

            typedef thrust::device_vector<int>::iterator Iterator; 
            thrust::reverse_iterator<Iterator> r_iter = thrust::make_reverse_iterator(d_vec + *curSize); // note that we point the iterator to the "end" of the device pointer area
            // thrust::copy(d_vec + (*curSize - n), d_vec + *curSize, ret_elements.begin());
            thrust::copy_n(r_iter,n,ret_elements.begin());
            
            if(isType == 2)
            {
                r_iter = thrust::make_reverse_iterator(d_values + *curSize);
                thrust::copy_n(r_iter,n,ret_elements.begin()+n);
            }

            *curSize -= n;
            return ret_elements;
        }

    };
    

    int main() {
        
        for(int lk = 0;lk<NRUNS;lk++)
        {
        	Heap hp;
		cout << "Run " << lk << "/" << (NRUNS - 1) << "-----" << endl;
            int elemSize = maxSize;
            // do{
            //     elemSize = getRandom(1,maxSize-hp.getSize());
            // }while(elemSize + hp.getSize() > maxSize);
            
            thrust::host_vector<int> elements(elemSize);
            thrust::host_vector<int> val(elemSize);
            FillArray(elements,elemSize);
            //FillArray(val,elemSize);
            //printArray(elements,elemSize);
            //printArray(val,elemSize);
            printf("No of Inserted Elements is = %d\n",elemSize);
            double starttime = rtclock(); 
            hp.insert(elements,elemSize);
            double endtime = rtclock(); 
            printtime("Insertion time: ", starttime, endtime); 

            printf("No of Deleted Elements is = %d\n",maxSizeDelete);
            starttime = rtclock();
            thrust::host_vector<int> res = hp.deleteElem(maxSizeDelete);
            endtime = rtclock();
            printtime("Delete: ", starttime, endtime);
            /*
	       for(int i = 0;i<6;i++)
                cout << res[i] <<", ";
		*/
	    cout << endl;
        }

        printf( " Over ");
        return 0;
    }
