#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

// Baseline kernel (your original)
__global__ void reduce0(int* input, int* output, int N) {
    extern __shared__ int sdata[];
    unsigned tid = threadIdx.x;
    unsigned i = blockIdx.x * blockDim.x + tid;
    
    sdata[tid] = (i < N) ? input[i] : 0;
    __syncthreads();

    for(int offset = blockDim.x/2; offset > 0; offset >>= 1) {
        if(tid < offset && (i + offset) < N) {
            sdata[tid] += sdata[tid + offset];
        }
        __syncthreads();
    }

    if(tid == 0) output[blockIdx.x] = sdata[0];
}

// Optimized kernel with warp shuffling and unrolling
template <unsigned blockSize>
__global__ void reduce3(int* input, int* output, int N) {
    extern __shared__ int sdata[];
    unsigned tid = threadIdx.x;
    unsigned i = blockIdx.x * (blockSize*2) + tid;
    
    // Load 2 elements per thread with stride
    sdata[tid] = 0;
    if(i < N) sdata[tid] = input[i];
    if(i + blockSize < N) sdata[tid] += input[i + blockSize];
    __syncthreads();

    // Unrolled block reduction
    if(blockSize >= 512) { if(tid < 256) sdata[tid] += sdata[tid + 256]; __syncthreads(); }
    if(blockSize >= 256) { if(tid < 128) sdata[tid] += sdata[tid + 128]; __syncthreads(); }
    if(blockSize >= 128) { if(tid < 64) sdata[tid] += sdata[tid + 64]; __syncthreads(); }

    // Warp-level reduction using shuffle
    if(tid < 32) {
        volatile int* vsdata = sdata;
        vsdata[tid] += vsdata[tid + 32];
        vsdata[tid] += vsdata[tid + 16];
        vsdata[tid] += vsdata[tid + 8];
        vsdata[tid] += vsdata[tid + 4];
        vsdata[tid] += vsdata[tid + 2];
        vsdata[tid] += vsdata[tid + 1];
    }

    if(tid == 0) output[blockIdx.x] = sdata[0];
}

int main(int argc, char** argv) {
    int k = 20;
    if(argc > 1) k = atoi(argv[1]);
    const int N = 1 << k;
    const int block_size = 512;
    
    // Host memory
    int* h_input = new int[N];
    int* h_output = new int[2]; // [reduce0, reduce3]
    for(int i=0; i<N; i++) h_input[i] = 1;

    // Device memory
    int *d_input, *d_intermediate;
    hipMalloc(&d_input, N*sizeof(int));
    hipMalloc(&d_intermediate, (N/block_size+1)*sizeof(int));
    hipMemcpy(d_input, h_input, N*sizeof(int), hipMemcpyHostToDevice);

    // Timing events
    hipEvent_t start0, stop0, start3, stop3;
    hipEventCreate(&start0); hipEventCreate(&stop0);
    hipEventCreate(&start3); hipEventCreate(&stop3);
    
    // Benchmark reduce0
    int current_N = N;
    hipEventRecord(start0);
    while(current_N > 1) {
        int grid_size = (current_N + block_size - 1) / block_size;
        reduce0<<<grid_size, block_size, block_size*sizeof(int)>>>(d_input, d_intermediate, current_N);
        current_N = grid_size;
    }
    hipEventRecord(stop0);
    hipEventSynchronize(stop0);
    hipMemcpy(&h_output[0], d_intermediate, sizeof(int), hipMemcpyDeviceToHost);

    // Reset intermediate data
    hipMemset(d_intermediate, 0, (N/block_size+1)*sizeof(int));

    // Benchmark reduce3
    current_N = N;
    hipEventRecord(start3);
    while(current_N > 1) {
        int grid_size = (current_N + block_size*2 - 1) / (block_size*2);
        reduce3<512><<<grid_size, block_size, block_size*sizeof(int)>>>(d_input, d_intermediate, current_N);
        current_N = grid_size;
    }
    hipEventRecord(stop3);
    hipEventSynchronize(stop3);
    hipMemcpy(&h_output[1], d_intermediate, sizeof(int), hipMemcpyDeviceToHost);

    // Print results
    float t0, t3;
    hipEventElapsedTime(&t0, start0, stop0);
    hipEventElapsedTime(&t3, start3, stop3);
    
    std::cout << "reduce0 sum: " << h_output[0] << " time: " << t0 << " ms\n";
    std::cout << "reduce3 sum: " << h_output[1] << " time: " << t3 << " ms\n";

    // Verification
    int cpu_sum = N;
    std::cout << "CPU sum: " << cpu_sum << std::endl;

    // Cleanup
    delete[] h_input;
    delete[] h_output;
    hipFree(d_input);
    hipFree(d_intermediate);
    return 0;
}
