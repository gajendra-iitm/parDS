#include "hip/hip_runtime.h"
#include<iostream>
#include<sys/time.h>
#include<hip/hip_runtime.h>
#include<vector>
#include<cstdlib>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/iterator/reverse_iterator.h>

#define maxSize 10000000
#define block_size 1024

using namespace std;


int map_size = 100000002;


struct cmap{
    int key;
    int value;
    int fill;
};


struct element_pair{
    int key;
    int value;
    int find;
};

struct element{
    int key;
    int find;
};

cmap *d_map, *h_map;



__global__ void kernel_index_to_fill(cmap *d_map, int *d_index_to_fill_in_hash_table, int *counter, int map_size){
    
     int tid = blockDim.x * blockIdx.x + threadIdx.x;                                              
     if(tid < map_size){
      
          if(d_map[tid].fill == 0){               
              
                int index = atomicInc((unsigned *)counter, maxSize);
                d_index_to_fill_in_hash_table[index] = tid;                
          }    
        
     }                     
    
}

__global__ void kernel_to_insert(cmap *d_map, element_pair *d_input, int *d_index_to_fill_in_hash_table, int insert_batch_size){


    int tid = blockDim.x * blockIdx.x + threadIdx.x;                                              
    if(tid < insert_batch_size && d_input[tid].find==0){
          d_map[d_index_to_fill_in_hash_table[tid]].key = d_input[tid].key;
          d_map[d_index_to_fill_in_hash_table[tid]].value = d_input[tid].value;
          d_map[d_index_to_fill_in_hash_table[tid]].fill = 1; 
    }
    
}


__global__ void search_kernel_pair(element_pair *d_search_input_pair, cmap *d_map, int a_size, int search_batch_size){

    
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(tid < search_batch_size * a_size){

        int index_of_element = tid % search_batch_size;
        int index_of_location = tid / search_batch_size;

        if(d_search_input_pair[index_of_element].key == d_map[index_of_location].key
           && d_search_input_pair[index_of_element].value == d_map[index_of_location].value && d_map[index_of_location].fill == 1){
            
            d_search_input_pair[index_of_element].find = 1;
        }
    }
}


__global__ void search_kernel_key(element *d_search_input_key, cmap *d_map, int a_size, int search_batch_size, int *d_count){

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < search_batch_size * a_size){

        int index_of_element = tid % search_batch_size;
        int index_of_location = tid / search_batch_size;

        if(d_search_input_key[index_of_element].key == d_map[index_of_location].key){

            int temp = atomicInc((unsigned int *)d_count, INT_MAX);
            //int temp = atomicInc((unsigned int *)d_count, -1);
            //d_search_input_pair[index_of_element].find = 1;

        }
    }

}

__global__ void fill_search_kernel_key(element *d_search_input_key, cmap *d_map, int a_size, int search_batch_size, element_pair *d_search_input_key_ans, int *index){

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < search_batch_size * a_size){

        int index_of_element = tid % search_batch_size;
        int index_of_location = tid / search_batch_size;

        if(d_search_input_key[index_of_element].key == d_map[index_of_location].key){
            int ind = atomicInc((unsigned int *)index, INT_MAX);
            //int ind = atomicInc((unsigned int *)index, -1);
            d_search_input_key_ans[ind].key = d_map[index_of_location].key;
            d_search_input_key_ans[ind].value = d_map[index_of_location].value;
        }
    }
}



__global__ void delete_kernel_pair(element_pair *d_delete_input_pair, cmap *d_map, int a_size, int delete_batch_size){


    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < delete_batch_size * a_size){

        int index_of_element = tid % delete_batch_size;
        int index_of_location = tid / delete_batch_size;

        if(d_delete_input_pair[index_of_element].key == d_map[index_of_location].key
           && d_delete_input_pair[index_of_element].value == d_map[index_of_location].value){

            d_map[index_of_location].key = 0;
            d_map[index_of_location].value = 0;
            d_map[index_of_location].fill = 0;
        }

    }

}

__global__ void delete_kernel(element *d_delete_input, cmap *d_map, int a_size, int delete_batch_size){


    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < delete_batch_size * a_size){

        int index_of_element = tid % delete_batch_size;
        int index_of_location = tid / delete_batch_size;

        if(d_delete_input[index_of_element].key == d_map[index_of_location].key){

            d_map[index_of_location].key = 0;
            d_map[index_of_location].value = 0;
            d_map[index_of_location].fill = 0;
        }

    }

}



class Map{

  public:
    

  public:

    Map(){

      
      //int map_element_counter = 0;

      
      h_map = (cmap*)malloc(map_size*sizeof(cmap));
      memset(h_map, 0, map_size*sizeof(cmap));

      hipMalloc(&d_map, map_size*sizeof(cmap));
      hipMemset(d_map, 0, map_size*sizeof(cmap));

    }

    void insertel(int insert_batch_size,  thrust::host_vector<thrust::host_vector<int>> &batch){        
        
      element_pair *d_input, *h_input;

      h_input = (element_pair*)malloc(insert_batch_size * sizeof(element_pair));

      for(int i=0 ; i<insert_batch_size ; i++){
          h_input[i].key = batch[i][0];
          h_input[i].value = batch[i][1];
          h_input[i].find = 0;
      }

      hipMalloc(&d_input, insert_batch_size*sizeof(element_pair));
      hipMemcpy(d_input, h_input, insert_batch_size*sizeof(element_pair), hipMemcpyHostToDevice);

      int block = ceil((float)(map_size*insert_batch_size)/(float)block_size);

      search_kernel_pair<<<block, block_size>>>(d_input, d_map, map_size, insert_batch_size);

      hipDeviceSynchronize();
      int *d_index_to_fill_in_hash_table, *counter;

      hipMalloc(&d_index_to_fill_in_hash_table, map_size*sizeof(int));
      hipMemset(d_index_to_fill_in_hash_table, 0, map_size*sizeof(int));

      hipMalloc(&counter, sizeof(int));
      hipMemset(counter, 0, sizeof(int));

      block = ceil((float)(map_size)/(float)block_size);

      kernel_index_to_fill<<<block, block_size>>>(d_map, d_index_to_fill_in_hash_table, counter, map_size);
      
      hipDeviceSynchronize();

      block = ceil((float)(insert_batch_size)/(float)block_size);

      kernel_to_insert<<<block, block_size>>>(d_map, d_input, d_index_to_fill_in_hash_table, insert_batch_size);

      hipDeviceSynchronize();

      hipMemcpy(h_map, d_map, map_size * sizeof(cmap), hipMemcpyDeviceToHost);

      
      // int count = 0;
      // for(int i=0 ; i<map_size ; i++){
      //     if(h_map[i].fill==1){
      //       cout<<"key = "<<h_map[i].key<<" "<<h_map[i].value<<"\n";
      //       count++;
      //     }
      // }

      // cout<<"Total Value1 = "<<count<<endl;
      // cout<<endl;


    }

    void searchPair(int search_batch_size, thrust::host_vector<thrust::host_vector<int>> &batch){

      element_pair *h_search_input_pair;
      element_pair *d_search_input_pair;


      h_search_input_pair = (element_pair*) malloc(search_batch_size * sizeof(element_pair));

      memset(h_search_input_pair, 0, search_batch_size*sizeof(element_pair));

      for(int i=0 ; i<search_batch_size ; i++){
          h_search_input_pair[i].key = batch[i][0];
          h_search_input_pair[i].value = batch[i][1];
      }

      hipMalloc(&d_search_input_pair, search_batch_size*sizeof(element_pair));
      hipMemcpy(d_search_input_pair, h_search_input_pair, search_batch_size*sizeof(element_pair), hipMemcpyHostToDevice);

                                //existing array Size * batch_size
      int sblock = ceil((float)(map_size*search_batch_size)/(float)block_size);

      search_kernel_pair<<<sblock, block_size>>>(d_search_input_pair, d_map, map_size, search_batch_size);

      hipMemcpy(h_search_input_pair, d_search_input_pair, search_batch_size*sizeof(element_pair), hipMemcpyDeviceToHost);

      //int flag = 1;

      // cout<<"\n\n\nFound value : \n";
      for(int i=0 ; i<search_batch_size ; i++){
          if(h_search_input_pair[i].find == 1){
            cout<<h_search_input_pair[i].key<<" "<<h_search_input_pair[i].value<<endl;
            // flag=0;
          }
      }

      cout<<endl;

    }


    void searchKey(int search_batch_size, thrust::host_vector<int> &keyvector){

      element *h_search_input_key;
      element *d_search_input_key;

      element_pair *h_search_input_key_ans;
      element_pair *d_search_input_key_ans;


      int *d_count, h_count, *d_index;


      hipMalloc(&d_count, sizeof(int));
      hipMemset(d_count, 0, sizeof(int));

      hipMalloc(&d_index, sizeof(int));
      hipMemset(d_index, 0, sizeof(int));

      h_search_input_key = (element*) malloc(search_batch_size * sizeof(element));

      memset(h_search_input_key, 0, search_batch_size*sizeof(element));

      for(int i=0 ; i<search_batch_size ; i++){
          h_search_input_key[i].key = keyvector[i];
      }

      hipMalloc(&d_search_input_key, search_batch_size*sizeof(element));
      hipMemcpy(d_search_input_key, h_search_input_key, search_batch_size*sizeof(element), hipMemcpyHostToDevice);


                                //existing array Size * batch_size
      int sblock = ceil((float)(map_size*search_batch_size)/(float)block_size);

      search_kernel_key<<<sblock, block_size>>>(d_search_input_key, d_map, map_size, search_batch_size, d_count);

      hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

      // cout<<"\nFound Entry with Keys = "<<h_count<<endl;

      h_search_input_key_ans = (element_pair*)malloc(h_count * sizeof(element_pair));
      memset(h_search_input_key_ans, 0, h_count*sizeof(element_pair));


      hipMalloc(&d_search_input_key_ans, h_count*sizeof(element_pair));
      hipMemset(d_search_input_key_ans, 0, h_count*sizeof(element_pair));


                                //existing array Size * batch_size
      sblock = ceil((float)(map_size*search_batch_size)/(float)block_size);

      fill_search_kernel_key<<<sblock, block_size>>>(d_search_input_key, d_map, map_size, search_batch_size, d_search_input_key_ans, d_index);



      hipMemcpy(h_search_input_key_ans, d_search_input_key_ans, h_count*sizeof(element_pair), hipMemcpyDeviceToHost);

      // cout<<"\n\n\nFound value with keys : \n";
      for(int i=0 ; i<h_count ; i++){
         
          cout<<h_search_input_key_ans[i].key<<" "<<h_search_input_key_ans[i].value<<endl;
       
          
      }

      
    }

    void deletePair(int delete_batch_size, thrust::host_vector<thrust::host_vector<int>> &batch){

      element_pair *h_delete_input_pair;
      element_pair *d_delete_input_pair;

      h_delete_input_pair = (element_pair*) malloc(delete_batch_size * sizeof(element_pair));

      memset(h_delete_input_pair, 0, delete_batch_size*sizeof(element_pair));

      for(int i=0 ; i<delete_batch_size ; i++){
          h_delete_input_pair[i].key = batch[i][0];
          h_delete_input_pair[i].value = batch[i][1];
      }

      hipMalloc(&d_delete_input_pair, delete_batch_size*sizeof(element_pair));
      hipMemcpy(d_delete_input_pair, h_delete_input_pair, delete_batch_size*sizeof(element_pair), hipMemcpyHostToDevice);


                                //existing array Size * batch_size
      int sblock = ceil((float)(map_size*delete_batch_size)/(float)block_size);

      delete_kernel_pair<<<sblock, block_size>>>(d_delete_input_pair, d_map, map_size, delete_batch_size);

      hipMemcpy(h_map, d_map, map_size * sizeof(cmap), hipMemcpyDeviceToHost);

      // cout<<"\n\n After Delettion:\n\n";
      // cout<<endl;
      // for(int i=0 ; i<map_size ; i++){
      //     if(h_map[i].fill==1)
      //       cout<<"key = "<<h_map[i].key<<" "<<h_map[i].value<<"\n";
      //     //cout<<h_map[i].value<<" ";
      // }



    }

    void deleteKey(int delete_batch_size, thrust::host_vector<int> &keyvector){

      element *h_delete_input;
      element *d_delete_input;

      h_delete_input = (element*) malloc(delete_batch_size * sizeof(element));

      memset(h_delete_input, 0, delete_batch_size*sizeof(element));

      for(int i=0 ; i<delete_batch_size ; i++){
          h_delete_input[i].key = keyvector[i];
      }

      hipMalloc(&d_delete_input, delete_batch_size*sizeof(element));
      hipMemcpy(d_delete_input, h_delete_input, delete_batch_size*sizeof(element), hipMemcpyHostToDevice);

                                //existing array Size * batch_size
      int sblock = ceil((float)(map_size*delete_batch_size)/(float)block_size);

      delete_kernel<<<sblock, block_size>>>(d_delete_input, d_map, map_size, delete_batch_size);

      hipMemcpy(h_map, d_map, map_size * sizeof(cmap), hipMemcpyDeviceToHost);

      // cout<<"\n\n After Delettion:\n\n";
      // cout<<endl;
      // for(int i=0 ; i<map_size ; i++){
      //     if(h_map[i].fill==1)
      //       cout<<"key = "<<h_map[i].key<<" "<<h_map[i].value<<"\n";
      //     //cout<<h_map[i].value<<" ";
      // }




    }




};


 int main(){

   Map mp;

   int insertSize = 10;
   thrust::host_vector<thrust::host_vector<int>> insertEl(insertSize, thrust::host_vector<int>(2,0));
    


   /*Start of Dummy Input*/
   insertEl[0][0] = 59;
   insertEl[0][1] = 6;
   // workDone[0] = 0;
   // searchDone[0] = 0;

   insertEl[1][0] = 59;
   insertEl[1][1] = 8;
   // workDone[1] = 0;
   // searchDone[1] = 0;

   insertEl[2][0] = 59;
   insertEl[2][1] = 9;
   // workDone[2] = 0;
   // searchDone[2] = 0;

   cout << "111\n";
   for(int i=3 ; i<insertSize ; i++){
       insertEl[i][0] = i;
       insertEl[i][1] = i+7;
       // workDone[i] = 0;	//This is important
       // searchDone[i] = 0;
   }
   // /*End of Dummy Input*/

  
   // //cout<<"size = "<<sizeof(insertEl)/sizeof(insertEl[0]);
   // int sz = sizeof(insertEl)/sizeof(insertEl[0]);
   int sz=10;
   mp.insertel(sz, insertEl);

   insertEl[0][0] = 58;
   insertEl[0][1] = 6;
   // workDone[0] = 0;
   // searchDone[0] = 0;

   insertEl[1][0] = 58;
   insertEl[1][1] = 8;
   // workDone[1] = 0;
   // searchDone[1] = 0;

   insertEl[2][0] = 58;
   insertEl[2][1] = 9;
   // workDone[2] = 0;
   // searchDone[2] = 0;

   cout << "222\n";
   for(int i=3 ; i<insertSize ; i++){
       insertEl[i][0] = i;
       insertEl[i][1] = i+7;
       // workDone[i] = 0;	//This is important
       // searchDone[i] = 0;
   }

   mp.insertel(sz, insertEl);

   sz = 3;
   vector<vector<int>> searchEl(sz, vector<int>(2,0));
    
   searchEl[0][0] = 58;
   searchEl[0][1] = 6;
   // workDone[0] = 0;
   // searchDone[0] = 0;

   searchEl[1][0] = 58;
   searchEl[1][1] = 8;
   // workDone[1] = 0;
   // searchDone[1] = 0;

   searchEl[2][0] = 58;
   searchEl[2][1] = 9;
  
   cout << "333\n";
   // for(int i=3 ; i<insertSize ; i++){
   //     insertEl[i][0] = i;
   //     insertEl[i][1] = i+7;
   //     // workDone[i] = 0;	//This is important
   //     // searchDone[i] = 0;
   // }

   // mp.search_pair(sz, searchEl);

   // sz = 2;

   // vector<int> skey(sz,0);

   // skey[0] = 58;
   // skey[1] = 3;

   // mp.search_key(sz, skey);

   // mp.delete_pair(sz, searchEl);

   sz = 2;

   cout << "555\n";
   thrust::host_vector<int> skey(sz,0);

   skey[0] = 3;
   skey[1] = 4;

   mp.deleteKey(sz, skey);

   cout << "444\n";
   skey[0] = 58;
   skey[1] = 59;

   mp.searchKey(sz, skey);

   cout << "666\n";

   return 0;
 }

